#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void bucket_initizalize(int *bucket){
  int i = threadIdx.x;
  bucket[i] = 0;
}

__global__ void bucket_add(int *key, int *bucket){
  int i = threadIdx.x;
  int num = key[i];
  atomicAdd(&bucket[num], 1);
}

__global__ void bucket_sort(int *key, int num, int length){
  int i = threadIdx.x;
  key[i+length] = num;
}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  bucket_initialize<<<1, range>>>(bucket);
  hipDeviceSynchronize();
  
  bucket_add<<<1, n>>>(key, bucket);
  hipDeviceSynchronize();
  
  int length = 0;
  for (int i=0; i<range; i++) {
    bucket_sort<<<1, bucket[i]>>>(key, i, length);
    lenght += bucket[i];
  }
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
}
